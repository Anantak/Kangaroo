#include "hip/hip_runtime.h"
#include "cu_gradient.h"

#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Image Gradient
//////////////////////////////////////////////////////

template<typename To, typename Ti>
__global__
void KernGradientMagnitude(Image<To> dOut, const Image<Ti> dIn)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    To v = (To)0.0;
    if(dOut.InBounds(x,y)) {
      v = dIn.template GetCentralDiffDx<To>(x,y) * dIn.template GetCentralDiffDx<To>(x,y); 
      v += dIn.template GetCentralDiffDy<To>(x,y) * dIn.template GetCentralDiffDy<To>(x,y);
      v = sqrt(v);
    }
    dOut(x,y) = v;
}

template<typename To, typename Ti>
void GradientMagnitude(Image<To> dOut, const Image<Ti> dIn)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, dOut);
    KernGradientMagnitude<<<gridDim,blockDim>>>(dOut,dIn);
}

// Explicit instantiation
template KANGAROO_EXPORT void GradientMagnitude<float,float>(Image<float>, const Image<float>);


} // namespace roo
